
#include <hip/hip_runtime.h>
// kernel_example.cu.cc
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "focal_loss_op.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/framework/register_types.h"

using namespace tensorflow;

// Define the CUDA kernel.
template <typename T>
__global__ void FocalLossGradCudaKernel(const int size, const T* in, T* out) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    out[i] = static_cast<T>(2) * ldg(in + i);
  }
}


template <typename T>
void FocalLossGradFunctorGPU<T>::operator()(const GPUDevice& d, typename TTypes<T>::ConstMatrix logits, typename TTypes<T>::ConstVec labels,
  typename TTypes<T>::Matrix softmax, typename TTypes<T>::Vec alpha, typename TTypes<T>::ConstScalar gamma, typename TTypes<T>::ConstVec focal_loss,
  typename TTypes<T>::Matrix grads) {
  // Launch the cuda kernel.
  //
  // See core/util/cuda_kernel_helper.h for example of computing
  // block count and thread_per_block count.
  int block_count = 1024;
  int thread_per_block = 20;
  FocalLossGradCudaKernel<T>
      <<<block_count, thread_per_block, 0, d.stream()>>>(1, nullptr, nullptr);
}

template struct FocalLossGradFunctorGPU<float>;

// #define DEFINE_GPU_SPECS(T)   \
//   template struct FocalLossGradFunctorGPU<T>;

// TF_CALL_GPU_NUMBER_TYPES(DEFINE_GPU_SPECS);

#endif  // GOOGLE_CUDA
